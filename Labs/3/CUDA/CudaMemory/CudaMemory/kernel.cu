
#include "hip/hip_runtime.h"

#include <iostream>


#include <stdio.h>

hipError_t calcDotProductWithCuda(int *c, const int *a, const int *b, unsigned int size);




int main()
{
    const int arraySize = 4;
    const int a[arraySize] = { 2, 2, 2, 2 };
    const int b[arraySize] = { 2, 2, 2, 2 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = calcDotProductWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int result = c[0] + c[1];

	std::cout << "Result of calculation:" << result << std::endl;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

#define THREADS_PER_BLOCK 2


__shared__ int blockNumbers[THREADS_PER_BLOCK];

__global__
void calcDotProdKernel(int * c, const int *b, const int * a)
{		

	int i = blockIdx.x * blockDim.x + threadIdx.x;		

	c[i] = a[i] * b[i];	
	printf("ThreadId: %d BlockId: %d using index: %d Calculated: %d \n", threadIdx.x, blockIdx.x, i, c[i]);

	blockNumbers[threadIdx.x] = c[i];
	

	//wait here until all threads reach this point inside the kernel
	__syncthreads();


	float subtotal = 0;
	for (int k = 0; k < blockDim.x; k++)
		subtotal += blockNumbers[k];

	c[blockIdx.x] = subtotal;
}





// Helper function for using CUDA to add vectors in parallel.
hipError_t calcDotProductWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    calcDotProdKernel<<<size/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_c, dev_b , dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
