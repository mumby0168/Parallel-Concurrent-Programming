#include "hip/hip_runtime.h"



#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "Ray.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "vec3.h"
#include <time.h>
#include <chrono>
#include <hip/hip_vector_types.h>


#define PARTICLE_COUNT 50

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;
using namespace std::chrono;

sphere spheres[PARTICLE_COUNT];
vec3 randoms[PARTICLE_COUNT];

hipArray *d_imageArray = 0;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const
	file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}
//__device__ vec3 castRay(const ray& r, const sphere *spheres) {
//	hit_record rec;
//	for (int i = 0; i < PARTICLE_COUNT; i++)
//	{
//		if (spheres[i].hit(r, 0.0, FLT_MAX)) {
//
//			//TODO: Pick a colour to render when hit
//			return vec3(0.5, 0.5, 0.5);
//		}
//		else {
//			//TODO: render the background colour
//			return vec3(1, 1, 1);
//		}
//	}
//	
//}

__device__ static int ticks = 1;

__device__ static float xStep = 0.01;
__device__ static float xPos = 0;

__device__ static float yStep = 0.01;
__device__ static float yPos = 0;

__device__ static float zStep = 0.001;
__device__ static float zPos = 0;


float generate_random() {

	auto r = rand() % 100;	
	if(rand() % 100 < 50)
		r = -r;
	return r / 1000.0;
}


extern "C" void init_particles()
{
	milliseconds ms = duration_cast<milliseconds>(
		system_clock::now().time_since_epoch());

	srand(ms.count());
}


void update_randoms() {
	for (int i = 0; i < PARTICLE_COUNT; i++) {
		randoms[i] = vec3(generate_random(), generate_random(), generate_random());
	}
}



__global__ void move_particles(sphere *spheres, const vec3 *randoms)
{	
	int i = threadIdx.x;	
	spheres[i].move(randoms[i].x(), randoms[i].y(), randoms[i].z());
}

__global__ void bound_particles(sphere *spheres)
{
	int i = threadIdx.x;
	int x = spheres[i].center.x();
	int y = spheres[i].center.y();
	int z = spheres[i].center.z();
	bool update = false;

	if (x > 1) {
		x = -1;
		update = true;
	}
	if (x < -1) {
		x = 1;
		update = true;
	}

	if (y > 1) {
		y = -1;
		update = true;
	}
	if (y< -1) {
		y= 1;
		update = true;
	}
	if (z > 1) {
		z = -1;
		update = true;
	}
	if (z < -1) {
		z = 1;
		update = true;
	}

	if(update)
		spheres[i].update_position(x, y, z);

}

__global__ void colour_particles()
{

}


////TODO: Possibly change this to check every particles boundaries.
//__global__ void create_world(hitable **d_list, hitable **d_world) {
//	
//	if (threadIdx.x == 0 && blockIdx.x == 0) {
//		*(d_list) = new sphere(vec3(xPos,yPos,zPos), 0.2);	
//		*d_world = new hitable_list(d_list, 1);
//	}
//}

__global__ void free_world(hitable **d_list, hitable **d_world) {
	delete *(d_list);
	delete *(d_list + 1);
	delete *d_world;
}

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_imageArray;
}

extern "C"
void freeTexture()
{
	checkCudaErrors(hipFreeArray(d_imageArray));
}


__global__ void
d_render(uchar4 *d_output, uint width, uint height, const sphere *spheres)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint i = y * width + x;
	

	float u = x / (float)width;
	float v = y / (float)height;
	u = 2.0*u - 1.0;
	v = -(2.0*v - 1.0);
	u *= width / height;
	u *= 2.0;
	v *= 2.0;
	
	if ((x < width) && (y < height))
	{		
		//for each pixel
		
		//fire a ray:
		ray r = ray(u, v);		
		for (int j = 0; j < PARTICLE_COUNT; j++)
		{				
			if (spheres[j].hit(r, 0.0, FLT_MAX))
			{				
				//TODO: This may not be best solution as a particle behind could be rendered first. i.e don't return.
				d_output[i] = spheres[j].color;		
				return;
			}			
		}
		d_output[i] = make_uchar4(124, 252, 0, 0);
	}			
}


// render image using CUDA
extern "C" 
void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 *output)
{
	sphere *d_spheres = 0;
	vec3 *d_randoms = 0;

	update_randoms();

	checkCudaErrors(hipMalloc((void **)&d_randoms, PARTICLE_COUNT * sizeof(vec3)));

	checkCudaErrors(hipMemcpy(d_randoms, randoms, PARTICLE_COUNT * sizeof(vec3), hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void **)&d_spheres, PARTICLE_COUNT * sizeof(sphere)));

	checkCudaErrors(hipMemcpy(d_spheres, spheres, PARTICLE_COUNT * sizeof(sphere), hipMemcpyHostToDevice));

	
	move_particles <<<1, 50 >>>(d_spheres, d_randoms);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	bound_particles << <1, 50 >> > (d_spheres);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	d_render << <gridSize, blockSize >> > (output, width, height, d_spheres);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(spheres, d_spheres, PARTICLE_COUNT * sizeof(sphere), hipMemcpyDeviceToHost));


	getLastCudaError("kernel failed");
}

#endif
