#include "hip/hip_runtime.h"



#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "Ray.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "vec3.h"
#include <time.h>
#include <chrono>
#include <hip/hip_vector_types.h>
#include "types.h"


#define PARTICLE_COUNT 50

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;
using namespace std::chrono;

sphere spheres[PARTICLE_COUNT];
vec3 randoms[PARTICLE_COUNT];

bool gravity_enabled = false;

hipArray *d_imageArray = 0;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const
	file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

extern "C" void set_gravity(bool value) {
	printf("gravity %d\n", value);
	gravity_enabled = value;
}


float generate_random() {

	float r = rand() % 100;	
	if(rand() % 100 < 50)
		r = -r;	
	return r / 100;		
}


extern "C" void init_particles()
{
	milliseconds ms = duration_cast<milliseconds>(
		system_clock::now().time_since_epoch());

	srand(ms.count());
}


void update_randoms() {
	for (int i = 0; i < PARTICLE_COUNT; i++) {
		randoms[i] = vec3(generate_random(), generate_random(), generate_random());
	}
}



__global__ void move_particles(sphere *spheres, const vec3 *randoms, const int *delta)
{	
	int i = threadIdx.x;
	float d = (*delta) / 1000.0;
	spheres[i].move(randoms[i].x() * d, randoms[i].y() * d, randoms[i].z() * d);
}

__global__ void apply_gravity(sphere *spheres, const int *delta)
{
	int i = threadIdx.x;
	float d = (*delta) / 1000.0;
	spheres[i].move(0,-0.9 * d,0);
}

__global__ void bound_particles(sphere *spheres)
{
	int i = threadIdx.x;
	int x = spheres[i].center.x();
	int y = spheres[i].center.y();
	int z = spheres[i].center.z();	
	bool update = false;

	if (x > 1) {
		x = -1;
		update = true;
	}
	if (x < -1) {
		x = 1;
		update = true;
	}

	if (y > 1) {
		y = -1;
		update = true;
	}
	if (y< -1) {
		y= 1;
		update = true;
	}
	if (z > 1) {
		z = -1;
		update = true;
	}
	if (z < -1) {
		z = 1;
		update = true;
	}

	if(update)
		spheres[i].update_position(x, y, z);

}


__global__ void colour_particles(const ColorMode *mode, sphere *spheres)
{
	int i = threadIdx.x;

	if (*mode == Solid) {
		spheres[i].solid_colour();
	}
	else if (*mode == CenterMass) {
		
	}
	else if (*mode == Speed) {

		float toRoute = pow((spheres[i].center.x() - spheres[i].previous_center.x()), 2) +
			pow((spheres[i].center.y() - spheres[i].previous_center.y()), 2) + pow((spheres[i].center.z() - spheres[i].previous_center.z()), 2);

		float distance = sqrt(toRoute);

		float percentage = (1.0 / distance) / 100;

		spheres[i].set_brightness(255 * percentage);
	}
}


////TODO: Possibly change this to check every particles boundaries.
//__global__ void create_world(hitable **d_list, hitable **d_world) {
//	
//	if (threadIdx.x == 0 && blockIdx.x == 0) {
//		*(d_list) = new sphere(vec3(xPos,yPos,zPos), 0.2);	
//		*d_world = new hitable_list(d_list, 1);
//	}
//}

__global__ void free_world(hitable **d_list, hitable **d_world) {
	delete *(d_list);
	delete *(d_list + 1);
	delete *d_world;
}

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_imageArray;
}

extern "C"
void freeTexture()
{
	checkCudaErrors(hipFreeArray(d_imageArray));
}


__global__ void
d_render(uchar4 *d_output, uint width, uint height, const sphere *spheres)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint i = y * width + x;		

	float u = x / (float)width;
	float v = y / (float)height;
	u = 2.0*u - 1.0;
	v = -(2.0*v - 1.0);
	u *= width / height;
	u *= 2.0;
	v *= 2.0;
	
	if ((x < width) && (y < height))
	{		
		//for each pixel
		
		//fire a ray:
		ray r = ray(u, v);		
		for (int j = 0; j < PARTICLE_COUNT; j++)
		{				
			if (spheres[j].hit(r, 0.0, FLT_MAX))
			{				
				//TODO: This may not be best solution as a particle behind could be rendered first. i.e don't return.
				d_output[i] = spheres[j].color;		
				return;
			}			
		}
		d_output[i] = make_uchar4(220, 220, 220, 255);
	}			
}


// render image using CUDA
extern "C" 
void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 *output, int deltaTime, ColorMode mode)
{
	sphere *d_spheres = 0;
	vec3 *d_randoms = 0;
	int *d_DeltaTime = 0;
	ColorMode *d_mode = 0;

	update_randoms();

	checkCudaErrors(hipMalloc((void **)&d_randoms, PARTICLE_COUNT * sizeof(vec3)));

	checkCudaErrors(hipMemcpy(d_randoms, randoms, PARTICLE_COUNT * sizeof(vec3), hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void **)&d_spheres, PARTICLE_COUNT * sizeof(sphere)));

	checkCudaErrors(hipMemcpy(d_spheres, spheres, PARTICLE_COUNT * sizeof(sphere), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void **)&d_DeltaTime, sizeof(int)));

	checkCudaErrors(hipMemcpy(d_DeltaTime, &deltaTime, sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void **)&d_mode, sizeof(ColorMode)));

	checkCudaErrors(hipMemcpy(d_mode, &mode, sizeof(ColorMode), hipMemcpyHostToDevice));

	
	move_particles<<<1, 50>>>(d_spheres, d_randoms, d_DeltaTime);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	if (gravity_enabled)
	{
		apply_gravity<<<1, 50 >>>(d_spheres, d_DeltaTime);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}

	bound_particles<<<1, 50>>>(d_spheres);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	colour_particles<<<1,50>>>(d_mode, d_spheres);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	d_render<<<gridSize, blockSize>>>(output, width, height, d_spheres);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(spheres, d_spheres, PARTICLE_COUNT * sizeof(sphere), hipMemcpyDeviceToHost));


	getLastCudaError("kernel failed");
}

#endif
