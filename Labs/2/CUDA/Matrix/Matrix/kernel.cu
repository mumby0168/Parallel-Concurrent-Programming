
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, unsigned int *blocks, unsigned int *threadsPerBlock);

void printArray(const int a[], int size);
void fillArray(int a[], int size);
void cleanUpMatrixOperation(int *pA, int *pB, int *pResult);
void addMatricesWithCuda(const int a[3][3], const int b[3][3], int c[3][3]);

__global__ 
void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	c[i] = a[i] + b[i];
}

__global__
void addMarticesKernel(int **resultMatrix, const int **matrixA, const int **matrixB)
{
	int x = threadIdx.x + (threadIdx.y * blockDim.x);
	int y = threadIdx.y;

	resultMatrix[x][y] = matrixA[x][y] + matrixB[x][y];
}

void cudaAddingExample()
{
	const int arraySize = 50;
	int a[arraySize];
	int b[arraySize];
	unsigned int blocks = 1;
	unsigned int threadsPerBlock = 10;

	fillArray(a, arraySize);
	fillArray(b, arraySize);

	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, &blocks, &threadsPerBlock);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");		
	}


	printArray(a, arraySize);
	printf("\n+\n ");
	printArray(b, arraySize);
	printf(" \n= \n");
	printArray(c, arraySize);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");		
	}

}

void cudaMatrixExample()
{

	int width = 3;
	int height = 3;

	const int a[3][3] = {
		{5, 5, 5},
		{10, 10, 10},
		{6, 6, 6}
	};

	const int b[3][3] = {
		{6, 5, 10},
		{6, 5, 10},
		{6, 5, 10}
	};

	int result[3][3] = {};
	


	addMatricesWithCuda(a, b, result);

}

void addMatricesWithCuda(const int a[][3], const int b[][3], int c[][3])
{
	hipError_t cudaStatus;
	int *pA = 0;
	int *pB = 0;
	int *pC = 0;

	//1. Setup device.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	//2. Allocate memory for 3 matrices
	int sizeOfMatrices = (3 * sizeof(int)) * 3;

	cudaStatus = hipMalloc((void**)&pA, sizeOfMatrices);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed allocating mem for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	cudaStatus = hipMalloc((void**)&pB, sizeOfMatrices);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed allocating mem for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	cudaStatus = hipMalloc((void**)&pC, sizeOfMatrices);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed allocating mem for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	//3. Copy memory from host structures to device.
	cudaStatus = hipMemcpy(pA, a, sizeOfMatrices, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed copying memory for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	cudaStatus = hipMemcpy(pB, b, sizeOfMatrices, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed copying memory for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	cudaStatus = hipMemcpy(pC, c, sizeOfMatrices, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed copying memory for matrix");
		cleanUpMatrixOperation(pA, pB, pC);
	}

	addMarticesKernel<<<1, dim3(3, 3) >>>((int**)pC, (const int**)&pA, (const int**)&pB);

	// 4. Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "add matrix launch failed: %s\n", hipGetErrorString(cudaStatus));
		cleanUpMatrixOperation(pA, pB, pC);
	}

	// 5. cudaDeviceSynchronize waits for the kernel to finish, and returns
	//	  any errors encountered during the launch?
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addMatrix!\n", cudaStatus);
		cleanUpMatrixOperation(pA, pB, pC);
	}

	
	cleanUpMatrixOperation(pA, pB, pC);
}

void cleanUpMatrixOperation(int *pA, int *pB, int *pResult)
{
	hipFree(pA);
	hipFree(pB);
	hipFree(pResult);
}

int main()
{
	cudaMatrixExample();
    return 0;
}

void printArray(const int a[], int size)
{
	printf("{");
	for (int i = 0; i < size; i++)
	{
		if (i % 50 == 0 && i != 0) {
			printf("\n");
		}
		printf("%d", a[i]);
		if (i != size -1)
		{
			printf(",");
		}
	}
	printf("}");
}

void fillArray(int a[], int size) 
{
	for (int i = 0; i < size; i += 2)
	{
		a[i] = 1;
		a[i + 1] = 2;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, unsigned int *blocks, unsigned int *threadsPerBlock)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	 	
	
	

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<5, 10>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
